#include "hip/hip_runtime.h"
/*
 * nvcc -I./ -I/usr/local/cuda/include -I/usr/local/cuda/lib64 -I../../../lib/cub/ -I../../../lib/cuda-samples/Common -lcudart -lcuda --std c++17 *.cu -o bubble_sort.exe
 * 
*/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

template<typename T>
struct ShouldSwap {
    /***
     * Sometimes the same functionality is needed in both the host 
     * and the device portions of CUDA code. To avoid code duplication, 
     * CUDA allows such functions to carry both host and device attributes, 
     * which means the compiler places one copy of that function into the 
     * host compilation flow (to be compiled by the host compiler, e.g. gcc or MSVC), 
     * and a second copy into the device compilation flow 
     * (to be compiled with NVIDIA’s CUDA compiler).
    */
    __host__ __device__ 
    virtual bool operator() (const T left, const T right) const;
};

template<typename T>
__host__ __device__ __inline__
void swap(T* a, T* r);

template<typename T>
__global__
void bubbleSort(T* v, const unsigned int n, ShouldSwap<T> shouldSwap);

int main(int argc, char** argv){
    const unsigned int size = 10;

    int h_v[size] = { 3, 7, 1, 10, 6, 9, 5, 2, 8, 4 };

    int *d_v = 0;
    
    hipMalloc((void**)&d_v, size * sizeof(int));

    hipMemcpy(d_v, h_v, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 grdDim(1, 1, 1);
    dim3 blkDim( size / 2, 1, 1);

    ShouldSwap<int> shouldSwap;

    bubbleSort<int> <<<grdDim, blkDim >>>(d_v, size, shouldSwap);
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipMemcpy(h_v, d_v, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_v);

    for (int i = 0; i < size; i++){
        std::cout << (i == 0 ? "{" : "") << h_v[i] << (i < size - 1 ? " ," : "}");
    }
    std::cout << std::endl;

    return 0;
}

template<typename T>
__host__ __device__
bool ShouldSwap<T>::operator() (const T left, const T right) const {
    return left > right;
}

template<typename T>
__host__ __device__ __inline__
void swap (T* a, T* b){
    T tmp = *a;
    *a = *b;
    *b = tmp;
}

template<typename T>
__global__
void bubbleSort(T* v, const unsigned int n, ShouldSwap<T> shouldSwao) {
    const unsigned int tIdx = threadIdx.x;

    for (unsigned int i = 0; i < n; i++){
        unsigned int offset = i % 2;
        unsigned int indiceGauche = 2 * tIdx + offset;
        unsigned int indiceDroite = indiceGauche + 1;

        if (indiceDroite < n) {
            if (shouldSwap(v[indiceGauche], v[indiceDroite])) {
                swap<T>(&v[indiceGauche], &v[indiceDroite]);
            }
        }

        __syncthreads();
    }
}